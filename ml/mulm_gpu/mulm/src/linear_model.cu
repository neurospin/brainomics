#include "hip/hip_runtime.h"
#include "linear_model.h"

/**kernel : computation of Y.X and the score for p permutations
Selection of the scores over the threshold.
**/
// Macro definition
#define CudaAssert( X ) if ( !(X) ) { return; }

__global__ void Kernel_DotProd(const flottant* d_X,
			const flottant* d_Y,
			const int* d_P,
			flottant *d_out,
			flottant *dalpha,
			int	Yoffset,
			size_t VectorSize,
			int num_vector_X,
			int num_vector_Y,
			int num_permut,
			unsigned int *cmpt,
			int dof,
			int threshold,
			int max_size,
			unsigned int *flag
			)
///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
{
	extern __shared__ flottant shared[];
	flottant *Y = &shared[0];
	flottant *X = &shared[TILE_WIDTH*blockDim.y];
	__shared__ int P[TILE_WIDTH*BLOCK_SIZEY];
	__shared__ flottant A[BLOCK_SIZEY];
	flottant results=0;
	flottant dot=0;
	int x_id = blockIdx.x;
	int y_id = blockIdx.y*blockDim.y + threadIdx.y;
	int z_id = blockIdx.z*blockDim.z + threadIdx.z;
	int intVectorSize = (VectorSize/TILE_WIDTH)*TILE_WIDTH;
	int restVectorSize = VectorSize%TILE_WIDTH;
	int ii, ind;
	int indx;
	CudaAssert( ( (*cmpt) < max_size ) );
	for(int k=0; k<VectorSize; k++)
		X[k]=d_X[x_id*VectorSize+k];
	X[VectorSize]=0;

	A[threadIdx.z]=dalpha[x_id*num_permut+z_id];

	for(ii=0;ii<intVectorSize;ii+=TILE_WIDTH)
	{
		for(int k=0; k<blockDim.y; k++)
			for(int j=0; j<TILE_WIDTH; j+=blockDim.y)
				Y[k*TILE_WIDTH + j + threadIdx.y]=d_Y[(blockIdx.y*blockDim.y + k)*VectorSize + ii + j + threadIdx.y];
		for(int k=0; k<blockDim.z; k++)
			for(int j=0; j<TILE_WIDTH; j+=blockDim.z)
				P[k*TILE_WIDTH + j + threadIdx.z]=d_P[(blockIdx.z*blockDim.z + k)*VectorSize + ii + j + threadIdx.z];
		__syncthreads();

	
		for(int j=0;j<TILE_WIDTH;j++)
		{
			ind=P[j+threadIdx.z*TILE_WIDTH];
			dot+= X[ind]*Y[j+threadIdx.y*TILE_WIDTH];
		}
		__syncthreads();
	}
	int j=0;
	ii=intVectorSize;
	if(restVectorSize)
	{
		for(int k=0; k<blockDim.y; k++)
				Y[k*restVectorSize+j+(threadIdx.y%restVectorSize)]=d_Y[(blockIdx.y*blockDim.y+k)*VectorSize+ii+j+(threadIdx.y%restVectorSize)];
		for(int k=0; k<blockDim.z; k++)
				P[k*restVectorSize+j+(threadIdx.z%restVectorSize)]=d_P[(blockIdx.z*blockDim.z+k)*VectorSize+ii+j+(threadIdx.z%restVectorSize)];
		__syncthreads();
		for(int j=0;j<restVectorSize;j++)
		{
			ind=P[j+threadIdx.z*restVectorSize];
			dot+= X[ind]*Y[j+threadIdx.y*restVectorSize];
		}
	}
	
	results = (dof*dot*dot)/(1-A[threadIdx.z]-dot*dot);
	if((abs(results) > threshold)){
		CudaAssert( ( *cmpt < max_size ) );
		indx = atomicInc(cmpt,num_vector_X*num_vector_Y*num_permut ); //valeur max : 2^31-1
		//indx = atomicAdd(cmpt,1);
		(*flag)|=(indx > max_size - 1) ;
		if(indx<max_size)
		{
		d_out[indx*4] = results;
		d_out[indx*4 + 1] = x_id;
		d_out[indx*4 + 2] = y_id;
		d_out[indx*4 + 3] = z_id;
	}}
}


///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/** Set CUDA environment, device, etc
Set block and grid size
Allocate, initialize and copy memory to device
Launch kernel.
**/

int  dotProdDevice(const flottant *x_in, 
			const flottant *y_in,
			const flottant *z_in,
			flottant *beta, 
			flottant *alpha,
			int *p_python,
			size_t VectorSize, 
			int num_vector_X,
			int num_vector_Y,
			int num_vector_Z,
			int num_permut,
			flottant divide,
			int threshold,
			int dev
			)
///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
{
	flottant *dx_in;
	flottant *dy_in;
	flottant *dalpha;
	int *dp_in;
	int *p_in;
	flottant *d_out;
	unsigned int *cmpt;
	unsigned int *flag;
	unsigned int hcmpt;
	unsigned int hflag;
	unsigned int total_cmpt = 0;
	size_t YSize = num_vector_Y*VectorSize*sizeof(flottant);
	size_t XSize = num_vector_X*VectorSize*sizeof(flottant);
	size_t PSize = num_permut*VectorSize*sizeof(int);
	size_t ASize = num_vector_X*num_permut*sizeof(flottant);
	size_t TotalSize = (num_vector_X*num_vector_Y)*(divide)*sizeof(flottant);
	size_t SubTotalMem;
/******************Get Device Properties of dev 0**************************/
	int deviceNb;
	hipGetDeviceCount(&deviceNb);
	if(dev<deviceNb)
		hipSetDevice(dev);
	else{
		printf("%d is not a valid device Id\n", dev);
		exit(-1);
	}
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, dev);
	size_t free, total;
	cutilSafeCall(hipMemGetInfo(&free, &total));
	size_t globalLeft = free*4/5;
	if(globalLeft< (XSize + YSize + PSize + ASize)){
		printf("Cannot allocate input data on device. Max Memory exceeded : %lu MB / %lu MB \n",(XSize + YSize + PSize + ASize)/1048576,globalLeft/1048576);
		exit(0);
	}
	else
		globalLeft-=(XSize + YSize + PSize + ASize);
		
    	printf("Total size of X, Y and P	=	%lu MB\n",(XSize + YSize + PSize)/1048576);
	printf("Global memory left after allocating X and Y	=	%lu MB\n",globalLeft/1048576);
	printf("Total global memory	 =	%lu MB\n",total/1048576);
	printf("Size needed for result array	=	 %lu MB\n",TotalSize/1048576);
 	
	int BlockSizeY = BLOCK_SIZEY;
	int BlockSizeP = BLOCK_SIZEY;
	int BlockSizeX = BLOCK_SIZEX;
		
/******************Allocation and Copy of input data**************************/
	
	int New_num_vector_Y=num_vector_Y;
	if(num_vector_Y%BlockSizeY)
		New_num_vector_Y = num_vector_Y + BlockSizeY-(num_vector_Y%BlockSizeY);
	size_t NewYSize = New_num_vector_Y*VectorSize*sizeof(flottant);
	
	int New_num_permut=num_permut;
	if(num_permut%BlockSizeP)
		New_num_permut = num_permut + BlockSizeP-(num_permut%BlockSizeP);
	size_t NewPSize = New_num_permut*VectorSize*sizeof(int);

	//p_in = (int*) realloc( p_in , NewPSize);
	p_in = (int*) malloc(NewPSize);
	for(int i=0; i<num_permut; i++)
		for (int j=0; j<VectorSize; j++)
			p_in[i*VectorSize+j] = p_python[i*VectorSize+j];
		
	for(int i=num_permut; i<New_num_permut; i++)
		for (int j=0; j<VectorSize; j++)
			p_in[i*VectorSize+j] = VectorSize;

	size_t NewASize = New_num_permut*num_vector_X*sizeof(flottant);
	//allocate memory for arrays in the device
	cutilSafeCall(hipMalloc( &dx_in, XSize));
	cutilSafeCall(hipMalloc( &dy_in, NewYSize));
	cutilSafeCall(hipMalloc( &dalpha, NewASize));
	cutilSafeCall(hipMalloc( &dp_in, NewPSize));

	cutilSafeCall(hipMalloc( &cmpt, sizeof(int)));
	cutilSafeCall(hipMalloc( &flag, sizeof(int)));
	
	//copy data from host to device
	cutilSafeCall(hipMemcpyAsync(dx_in, x_in, XSize, hipMemcpyHostToDevice));
	cutilSafeCall(hipMemcpyAsync(dy_in, y_in, YSize, hipMemcpyHostToDevice));
	cutilSafeCall(hipMemset(&dy_in[num_vector_Y*VectorSize], 0 , (NewYSize- YSize)));
	cutilSafeCall(hipMemcpyAsync(dalpha, alpha, ASize, hipMemcpyHostToDevice));
	cutilSafeCall(hipMemset(&dalpha[num_permut*num_vector_X], 0 , (NewASize - ASize)));

	cutilSafeCall(hipMemcpyAsync(dp_in, p_in, NewPSize, hipMemcpyHostToDevice));
	//cutilSafeCall(hipMemset(&dp_in[num_permut*VectorSize], 0 , (NewPSize - PSize)));
	
	int maxGridSize0 = deviceProp.maxGridSize[0];
	int maxGridSize1 = deviceProp.maxGridSize[1];
	int maxGridSize2 = deviceProp.maxGridSize[2];
	//printf("maxGridSize  = %d\n",deviceProp.maxGridSize[0]);
	int nb_block_x = num_vector_X;
	int nb_block_y = New_num_vector_Y/BlockSizeY;
	int nb_block_p = New_num_permut/BlockSizeP;
	//size_t size = num_vector_X*New_num_vector_Y*sizeof(flottant);

	if(nb_block_x>maxGridSize0)
	{
		printf("Exceeding Max X Grid Size \n");
		nb_block_x = nb_block_x/(nb_block_x/maxGridSize0+1)+1;
	}
	if(nb_block_y>maxGridSize1)
	{
		printf("Exceeding Max Y Grid Size \n");
		nb_block_y = nb_block_y/(nb_block_y/maxGridSize1+1)+1;
	}
	if(nb_block_p>maxGridSize2)
	{
		printf("Error : Exceeding Max Z Grid Size \nCannot have num_permut bigger than %d\n",maxGridSize2);
		exit(0);
	}
	
	size_t size = (nb_block_x*BlockSizeX)*(nb_block_y*BlockSizeY)*(divide)*sizeof(flottant);

	if(size>globalLeft)
	{
		printf("Exceeding Memory Capacity \n");
		printf("current nb_blocks=%d, size=%lu B\n",nb_block_y,size);
		nb_block_y = nb_block_y/(size/globalLeft+1)+1;
		printf("size/globalLeft+1=%lu, new nb_block=%d\n",size/globalLeft+1,nb_block_y);
	}

	int XVectPerCall=nb_block_x*BlockSizeX;
	int YVectPerCall=nb_block_y*BlockSizeY;
	printf("number of Xvectors per call = %d\nnumber of Yvectors per call = %d\n",XVectPerCall,YVectPerCall);
	int nbXVect,nbYVect;

	SubTotalMem = (XVectPerCall*YVectPerCall)*(divide)*sizeof(flottant);
	//printf("d_out memory allocated = %lu\n", SubTotalMem);
	cutilSafeCall(hipMalloc( &d_out, SubTotalMem));
	int dof=VectorSize - 1 - num_vector_Z;
	int pt_vector_X=0, pt_vector_Y=0;
	dim3 dimBlock(BlockSizeX, BlockSizeY, BlockSizeP);
	dim3 dimGrid(nb_block_x, nb_block_y, nb_block_p);
	
/********************kernel launches**************************/
	while (pt_vector_X<num_vector_X){
		nbXVect = XVectPerCall;
		if ( pt_vector_X + XVectPerCall > num_vector_X)
			nbXVect = num_vector_X - pt_vector_X;
		pt_vector_Y=0;
		
		while(pt_vector_Y<New_num_vector_Y){
			nbYVect = YVectPerCall;
			if ( pt_vector_Y + YVectPerCall > New_num_vector_Y)
				nbYVect = New_num_vector_Y - pt_vector_Y;
			printf("Number of vectors treated in this loop = %dx%d\n", nbXVect,nbYVect);
			printf("##############\n");
			printf("Number of threads	=	%dx%dx%d \n"\
				 "Number of blocks	=	%dx%dx%d \n"\
				 ,dimBlock.x, dimBlock.y, dimBlock.z, dimGrid.x, dimGrid.y, dimGrid.z);
			printf("##############\n");

			cutilSafeCall(hipMemset(d_out, 0 , SubTotalMem));
			cutilSafeCall(hipMemset( cmpt, 0 , sizeof(int)));
			cutilSafeCall(hipMemset( flag, 0 , sizeof(int)));
			size_t max_size = (XVectPerCall*YVectPerCall)*(divide/4);
			printf("Maximum number of values allowed = %G\n", (double)max_size);
			size_t sharedMem = (VectorSize + 1 + TILE_WIDTH*BlockSizeY)*sizeof(flottant);
			Kernel_DotProd<<< dimGrid , dimBlock, sharedMem>>>(&dx_in[pt_vector_X*VectorSize], &dy_in[pt_vector_Y*VectorSize], dp_in, d_out,dalpha, pt_vector_Y, VectorSize, num_vector_X, New_num_vector_Y,New_num_permut,cmpt,dof,threshold, max_size, flag); 
			cutilSafeCall(hipGetLastError());
			cutilSafeCall(hipMemcpy(&hcmpt, cmpt, sizeof(int), hipMemcpyDeviceToHost));
			cutilSafeCall(hipMemcpy(&hflag, flag, sizeof(int), hipMemcpyDeviceToHost));
			printf("flag = %d\n", hflag);
			if(hflag!=0){
				 printf("############\nWARNING");
				 printf(" : Beta size limit was reached. Some values were not calculated.\nTry tuning the output size with the \"divide\" parameter.\n");
				 printf("############\n");
				 hcmpt=max_size*80/100;
			}
			pt_vector_Y += YVectPerCall;
//			printf("Max number of values =  %G\nnumber of values calculated in this launch : %d\n", (double)max_size,hcmpt);
			cutilSafeCall(hipMemcpy(&beta[4*total_cmpt], d_out, hcmpt*4*sizeof(flottant), hipMemcpyDeviceToHost));
			total_cmpt += hcmpt;
		}
		pt_vector_X += XVectPerCall;
	}
	printf("number of calculated values =	%G\n",(double)total_cmpt);

	printf("DotProd done\n");
	// cleanup
	hipFree(p_in);
	hipFree (dx_in);
	hipFree (dy_in);
	hipFree (dp_in);
	hipFree (d_out);

	return total_cmpt;
}

/**Main for mulm regression called by run_mulm_permut.py
**/
int MULMRegression(flottant* X, int size_X,
		   flottant* Y, int size_Y,
		   flottant* Z, int size_Z,
		   int* P, int size_permut,
		   int VectorSize,
		   flottant divide,
		   int threshold,
		   flottant* beta, int size_B,
		   int dev
		   )
{
	int num_vector_X=size_X/VectorSize;
	int num_vector_Y=size_Y/VectorSize;
	int num_vector_Z=size_Z/VectorSize;
	int num_permut=size_permut/VectorSize;

	
	printf("Arguments : \n");
	printf("num_vector_X/Y/Z : %d, %d, %d\n",num_vector_X,num_vector_Y,num_vector_Z);
	printf("num_permut : %d\nVectorSize : %d\nThreshold : %d\n",num_permut,VectorSize,threshold);
		
	int lost_dof=0;
	unsigned int values = 0;
//step 1 (regression 1): extract Covariables effect from y
	lost_dof+=orthoNorm(Z, VectorSize, num_vector_Z);
	
	normalize(Y, VectorSize, num_vector_Y);

	proj(Y, Z, VectorSize, num_vector_Y, num_vector_Z);
	normalize(Y, VectorSize, num_vector_Y);

// step 2 (regression 2): extract effect of z from x
	normalize(X, VectorSize, num_vector_X);

	proj(X, Z, VectorSize, num_vector_X, num_vector_Z);
	normalize(X, VectorSize, num_vector_X);

	flottant *alpha=(flottant*)calloc(num_vector_Y*num_permut,sizeof(flottant));
	if(alpha==NULL){
		printf("Error : couldn't allocate alpha\n");
		exit(0);
	}
	beta=(flottant*)malloc(size_B*sizeof(flottant));
	if(beta==NULL){
		printf("Error : couldn't allocate beta\n");
		exit(0);
	}
//  step 4: original regression (3)
	dotProdPerm(Y, Z, P, alpha, VectorSize, num_vector_Y, num_vector_Z, num_permut);
	values = dotProdDevice(Y,X,Z,beta,alpha,P,VectorSize,num_vector_Y, num_vector_X,num_vector_Z,num_permut,divide,threshold,dev);

#ifdef DISPLAY
	SaveMat(beta,4,values, "beta");
	flottant *mean_scores = (flottant*)calloc(num_permut*4,sizeof(flottant));
	flottant *count = (flottant*)calloc(num_permut,sizeof(flottant));
	for(int i=0; i<num_permut*4; i++)
		mean_scores[i]=0;
// on prend le beta max pour chaque permut		
	for(int i=0; i<values; i++){
		if(beta[4*i] > mean_scores[(int)beta[4*i+3]*4+1]){
			mean_scores[(int)beta[4*i+3]*4+1] = beta[4*i];
			mean_scores[(int)beta[4*i+3]*4+2] = beta[4*i+1];
			mean_scores[(int)beta[4*i+3]*4+3] = beta[4*i+2];
		}
//on prend la moyenne des betas pour une permut
		mean_scores[(int)beta[4*i+3]*4]+=beta[4*i];
		count[(int)beta[4*i+3]]++;
	}
	for(int i=0; i<num_permut; i++)
		mean_scores[i*4]/=count[i];
	SaveMat(mean_scores,4,num_permut,"mean_scores");
#endif
	free(alpha);	
	printf("end of MulmRegression call\n");
	free(beta);
	return values;
}


