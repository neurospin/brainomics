#include "hip/hip_runtime.h"
#include "linear_model.h"
/**
   kernel : computation of Y.X and the score for p permutations
**/
// Macro definition
#define CudaAssert( X )  if ( !(X) ) { return; }

#define	BLOCK_SIZEX	1

#define	BLOCK_SIZEY 32
#define bsize 64
#define nb_permut_per_th 8

int get_block_sizey() { return BLOCK_SIZEY; }
int get_bsize() { return bsize; }
int get_nb_permut_per_th() { return nb_permut_per_th; }

/**
 * Cuda kernel for d_X.d_y dot product and F scores with permutations
 * and covariates bias correction, plus thresholding for sparse COO
 * array output.
 **/
__global__ void Kernel_DotProd(const flottant* d_X,
			       const flottant* d_Y,
			       const int* d_P,
			       double *d_out,
			       const size_t VectorSize,
			       const int num_vector_Y,
			       const int num_vector_X)
{
  extern __shared__ flottant shared[];
  flottant *X = &shared[0];
  __shared__ double dot[nb_permut_per_th*bsize];
  flottant Y;
  int tid= threadIdx.z+threadIdx.y*blockDim.z;
  int y_id = blockIdx.y*blockDim.y + threadIdx.y;
  int z_id = blockIdx.z*blockDim.z + threadIdx.z;
  unsigned int ind, k;
  double b2;
  // the block of threads loads X
  for(int i=0; i<num_vector_X; i++) {

    // initialize dot[] to 0
    for(int j=0;j<nb_permut_per_th;j++)
      dot[j*bsize+tid]=0;

    for(k=0; k<VectorSize; k+=blockDim.z*blockDim.y)
      if((k+threadIdx.y+threadIdx.z*blockDim.y)<VectorSize)
	X[k+threadIdx.y+threadIdx.z*blockDim.y]=d_X[(blockIdx.x+i)*VectorSize+k+threadIdx.y+threadIdx.z*blockDim.y];

    __syncthreads(); // wait until data are loaded
  
    // dot(X[perm],Y), ie. beta computation
    for(k=0;k<VectorSize;k++)
      {
	Y=d_Y[num_vector_Y*(k)+y_id];
	for(int j=0;j<nb_permut_per_th;j++)
	  {
	    ind=d_P[(nb_permut_per_th*z_id+j)*VectorSize+k];
	    dot[j*bsize+tid]+= X[ind]*Y;
	  }
      }
    
    for(int j=0;j<nb_permut_per_th;j++) {
      b2 = dot[j*bsize+tid] * dot[j*bsize+tid];
      d_out[(j+nb_permut_per_th*z_id)* num_vector_Y + y_id] += b2;

    }
	
  }
}

/**
 * Management of Cuda host code and kernel Launch
 *
 * 1 - Set CUDA environment, device, etc 
 * 2 - Set block and grid size
 * 3 - Allocate, initialize and copy memory to device 
 * 4 - Launch kernel
 **/
void dotProdDevice(const flottant *x_in, 
		   const flottant *y_in,
		   double *beta,
		   int *p_python,
		   size_t VectorSize,
		   int num_vector_X,
		   int num_vector_Y,
		   int num_permut,
		   int dev
		   )
{
  flottant *dx_in;
  flottant *dy_in;
  int *dp_in;
  double *d_out;
  size_t YSize = num_vector_Y * VectorSize * sizeof(flottant);
  size_t XSize = num_vector_X * VectorSize * sizeof(flottant);
  size_t PSize = num_permut * VectorSize * sizeof(int);
  size_t SubTotalMem;

  //////////////// Get Properties of the device and parametrize tiling ////////////////
  int deviceNb;
  hipGetDeviceCount(&deviceNb);
  hipSetDeviceFlags(hipDeviceScheduleYield);
  if(dev<deviceNb) {
    hipSetDevice(dev);
  } else {
    fprintf(stderr, "# %d is not a valid device Id\n", dev);
    exit(-1);
  }
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, dev);
  size_t free_device_mem, total_device_mem;
  hipMemGetInfo(&free_device_mem, &total_device_mem);
  size_t globalLeft = free_device_mem * 4 / 5;
  if(globalLeft < (XSize + YSize + PSize)) {
    fprintf(stderr, "# Cannot allocate input data on device. Max Memory exceeded : %lu MB / %lu MB \n",
	    (XSize + YSize + PSize) / 1048576, globalLeft / 1048576);
    exit(0);
  } else 
    globalLeft -= (XSize + YSize + PSize);
 	
  int BlockSizeY = BLOCK_SIZEY;
  int BlockSizeP = bsize / BLOCK_SIZEY;
  int BlockSizeX = BLOCK_SIZEX;
	
  int maxGridSize1 = deviceProp.maxGridSize[1];
  int maxGridSize2 = deviceProp.maxGridSize[2];

  int nb_block_x = 1; // num_vector_X;
  int nb_block_y = num_vector_Y / BlockSizeY;
  int nb_block_p = num_permut / BlockSizeP;

  if(nb_block_y > maxGridSize1) {
    fprintf(stderr, "# Exceeding Max Y Grid Size \n");
    nb_block_y = nb_block_y / (nb_block_y / maxGridSize1 + 1) + 1;
  }
  if(nb_block_p > maxGridSize2) {
    fprintf(stderr, "# Error : Exceeding Max Z Grid Size \nCannot have num_permut bigger than %d\n",maxGridSize2);
    exit(0); 
  }

   size_t size = nb_block_y * BlockSizeY * nb_block_p * BlockSizeP * sizeof(flottant);

  if(size > globalLeft) {
    fprintf(stderr, "# Exceeding GPU Memory Capacity \n");
    nb_block_y = nb_block_y / (size / globalLeft + 1) + 1;
    fprintf(stderr, "# Number of GPU iterations needed =  %lu\n", size / globalLeft + 1);
  }

  int XVectPerCall = nb_block_x * BlockSizeX;
  int YVectPerCall = nb_block_y * BlockSizeY;

  dim3 dimBlock(BlockSizeX, BlockSizeY, BlockSizeP);
  dim3 dimGrid(nb_block_x, nb_block_y, nb_block_p / nb_permut_per_th);

  SubTotalMem = YVectPerCall * nb_block_p * BlockSizeP * sizeof(double);
  fprintf(stderr, "# d_out memory allocated = %lu\n", SubTotalMem);

  size_t sharedMem = ((VectorSize / 32) * 32) * sizeof(flottant);
  fprintf(stderr,"# SharedMem = %G\n", (double)sharedMem);

  ///////////////// Allocation and Copy of input data ////////////////

  // allocate memory for arrays in the device
 hipMalloc( &dx_in, XSize);
 hipMalloc( &dy_in, YSize);
 hipMalloc( &dp_in, PSize);
	
  //copy data from host to device
  hipMemcpyAsync(dx_in, x_in, XSize, hipMemcpyHostToDevice);
  hipMemcpyAsync(dy_in, y_in, YSize, hipMemcpyHostToDevice);
  hipMemcpyAsync(dp_in, p_python, PSize, hipMemcpyHostToDevice);
  hipMalloc(&d_out, SubTotalMem);
  	
  ///////////////// kernel launches ///////////////// 

  int pt_vector_X = 0, pt_vector_Y = 0;
  int loop_it = 1, nbXVect = 0, nbYVect = 0;

  while (pt_vector_X < num_vector_X) {
    nbXVect = XVectPerCall;
    if ( pt_vector_X + XVectPerCall > num_vector_X)
      nbXVect = num_vector_X - pt_vector_X;
    pt_vector_Y=0;
		
    while(pt_vector_Y < num_vector_Y) {
      nbYVect = YVectPerCall;
      if (pt_vector_Y + YVectPerCall > num_vector_Y)
	nbYVect = num_vector_Y - pt_vector_Y;

      fprintf(stderr, "# Iteration No %d, blocks: %dx%dx%d, threads: %dx%dx%d \n",
	      loop_it, dimGrid.x, dimGrid.y, dimGrid.z, dimBlock.x, dimBlock.y, dimBlock.z);

      hipMemset(d_out, 0 , SubTotalMem);

      //  hipFuncSetCacheConfig(reinterpret_cast<const void*>(Kernel_DotProd),hipFuncCachePreferL1);
      Kernel_DotProd<<< dimGrid , dimBlock, sharedMem>>>(&dx_in[pt_vector_X*VectorSize], 
							 &dy_in[pt_vector_Y*VectorSize], 
							 dp_in, d_out,
							 VectorSize, num_vector_Y,
							 num_vector_X); 
      hipGetLastError();
      pt_vector_Y += YVectPerCall;
      // retrieve scores for the iteration
      hipMemcpy(beta, d_out, SubTotalMem, hipMemcpyDeviceToHost);
      loop_it++;
    }
    pt_vector_X += num_vector_X;
  }
  
  // cleanup
  hipFree(dx_in);
  hipFree(dy_in);
  hipFree(dp_in);
  hipFree(d_out);
}

/**
   Main for mulm regression
**/
int OLSRegression(flottant* X, int size_X,
		   flottant* Y, int size_Y,
		   int* P, int size_permut,
		   double* beta, int size_B,
		   int VectorSize,
		   int dev
		   )
{
  int num_vector_X = size_X / VectorSize;
  int num_vector_Y = size_Y / VectorSize;
  int num_permut = size_permut / VectorSize;

  dotProdDevice(X,Y,beta,P,VectorSize,
		num_vector_X, num_vector_Y, num_permut, dev);
  return 0;
}


