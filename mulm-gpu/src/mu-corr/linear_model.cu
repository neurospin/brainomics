#include "hip/hip_runtime.h"
#include "linear_model.h"
/**
   kernel : computation of Y.X and the score for p permutations
   Selection of the scores over the threshold.
**/
// Macro definition
#define CudaAssert( X )  if ( !(X) ) { return; }

#define	BLOCK_SIZEX	1

#define	BLOCK_SIZEY 32
#define bsize 128
#define nb_permut_per_th 8

int get_block_sizey() { return BLOCK_SIZEY; }
int get_bsize() { return bsize; }
int get_nb_permut_per_th() { return nb_permut_per_th; }

/**
 * Cuda kernel for d_X.d_y dot product and F scores with permutations
 * and covariates bias correction, plus thresholding for sparse COO
 * array output.
 **/
__global__ void Kernel_DotProd(const flottant* d_X,
			       const flottant* d_Y,
			       const int* d_P,
			       flottant *d_out,
			       const flottant *dalpha,
			       const size_t VectorSize,
			       const int num_vector_Y,
			       const int num_permut,
			       unsigned int *cmpt,
			       const int dof,
			       const double threshold,
			       const int max_size,
			       unsigned int *flag
			       )
{
  extern __shared__ flottant shared[];
  flottant *X = &shared[0];
  flottant A;
  int tid= threadIdx.z+threadIdx.y*blockDim.z;
  __shared__ double dot[nb_permut_per_th*bsize];
  int y_id = blockIdx.y*blockDim.y + threadIdx.y;
  int z_id = blockIdx.z*blockDim.z + threadIdx.z;
  unsigned int ind,k;
  
  // initialize dot[] to 0
  for(int j=0;j<nb_permut_per_th;j++)
    dot[j*bsize+tid]=0;

  // the block of threads loads X
  for(k=0; k<VectorSize; k+=blockDim.z*blockDim.y)
    if((k+threadIdx.y+threadIdx.z*blockDim.y)<VectorSize)
      X[k+threadIdx.y+threadIdx.z*blockDim.y]=d_X[blockIdx.x*VectorSize+k+threadIdx.y+threadIdx.z*blockDim.y];

  __syncthreads(); // wait until data are loaded
  
  // dot(X[perm],Y), ie. beta computation
  for(k=0;k<VectorSize;k++)
    {
      A=d_Y[num_vector_Y*(k)+y_id];
      for(int j=0;j<nb_permut_per_th;j++)
	{
	  ind=d_P[(nb_permut_per_th*z_id+j)*VectorSize+k];
	  dot[j*bsize+tid]+= X[ind]*A;
	}
    }

  // F score and filter
  for(int j=0;j<nb_permut_per_th;j++)
    {
      // load a^2 for covariates bias correction
      A=dalpha[y_id+num_vector_Y*(nb_permut_per_th*z_id+j)];
      /* compute F score*/
      dot[bsize*j+tid]*=dot[bsize*j+tid]; // b^2
      A+=dot[bsize*j+tid]; // a^2 + b^2 
      dot[bsize*j+tid] *= dof; // b^2 * dof
      dot[bsize*j+tid]/=(1.-A); // (b^2 * dof) / (1 - (a^2 + b^2))
      
      if(dot[bsize*j+tid] > threshold )
	{
	  // reach max allocated size?
	  CudaAssert( ( *cmpt < max_size ) );
	  // atomic incrementation to write next element in d_out
	  ind = atomicInc(cmpt, max_size);
	  // set a flag if max_size was reached
	  (*flag)|=(ind > max_size - 1) ;
	  if(ind<max_size)
	    {
	      d_out[4 * ind] = dot[bsize*j+tid];
	      d_out[4 * ind + 1] = blockIdx.x;
	      d_out[4 * ind +2] = y_id;
	      d_out[4 * ind +3] = nb_permut_per_th*z_id+j;
	    }
	}
    }
}

/**
 * Management of Cuda host code and kernel Launch
 *
 * 1 - Set CUDA environment, device, etc 
 * 2 - Set block and grid size
 * 3 - Allocate, initialize and copy memory to device 
 * 4 - Launch kernel
 **/
int  dotProdDevice(const flottant *x_in, 
		   const flottant *y_in,
		   flottant *beta, 
		   flottant *alpha,
		   int *p_python,
		   size_t VectorSize, int nSubj, 
		   int num_vector_X,
		   int num_vector_Y,
		   int num_vector_Z,
		   int num_permut,
		   flottant divide,
		   double threshold,
		   int dev
		   )
{
  flottant *dx_in;
  flottant *dy_in;
  flottant *dalpha;
  int *dp_in;
  flottant *d_out;
  unsigned int *cmpt;
  unsigned int *flag;
  unsigned int hcmpt;
  unsigned int hflag;
  unsigned int total_cmpt = 0;
  size_t YSize = num_vector_Y * VectorSize * sizeof(flottant);
  size_t XSize = num_vector_X * VectorSize * sizeof(flottant);
  size_t PSize = num_permut * VectorSize * sizeof(int);
  size_t ASize = num_vector_Y * num_permut * sizeof(flottant);
  size_t TotalSize = num_vector_X * num_vector_Y * divide * sizeof(flottant);
  size_t SubTotalMem;

  //////////////// Get Properties of the device and parametrize tiling ////////////////
  int deviceNb;
  hipGetDeviceCount(&deviceNb);
  hipSetDeviceFlags(hipDeviceScheduleYield);
  if(dev<deviceNb) {
    hipSetDevice(dev);
  } else {
    fprintf(stderr, "# %d is not a valid device Id\n", dev);
    exit(-1);
  }
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, dev);
  size_t free_device_mem, total_device_mem;
  hipMemGetInfo(&free_device_mem, &total_device_mem);
  size_t globalLeft = free_device_mem * 4 / 5;
  if(globalLeft < (XSize + YSize + PSize + ASize)) {
    fprintf(stderr, "# Cannot allocate input data on device. Max Memory exceeded : %lu MB / %lu MB \n",
	    (XSize + YSize + PSize + ASize) / 1048576, globalLeft / 1048576);
    exit(0);
  } else 
    globalLeft -= (XSize + YSize + PSize + ASize);
 	
  int BlockSizeY = BLOCK_SIZEY;
  int BlockSizeP = bsize / BLOCK_SIZEY;
  int BlockSizeX = BLOCK_SIZEX;
	
  int maxGridSize0 = deviceProp.maxGridSize[0];
  int maxGridSize1 = deviceProp.maxGridSize[1];
  int maxGridSize2 = deviceProp.maxGridSize[2];

  int nb_block_x = num_vector_X;
  int nb_block_y = num_vector_Y / BlockSizeY;
  int nb_block_p = num_permut / BlockSizeP;

  if(nb_block_x > maxGridSize0) {
    fprintf(stderr, "# Exceeding Max X Grid Size \n");
    nb_block_x = nb_block_x / (nb_block_x / maxGridSize0 + 1) + 1;
  }
  if(nb_block_y > maxGridSize1) {
    fprintf(stderr, "# Exceeding Max Y Grid Size \n");
    nb_block_y = nb_block_y / (nb_block_y / maxGridSize1 + 1) + 1;
  }
  if(nb_block_p > maxGridSize2) {
    fprintf(stderr, "# Error : Exceeding Max Z Grid Size \nCannot have num_permut bigger than %d\n",maxGridSize2);
    exit(0); 
  }

   size_t size = nb_block_x * BlockSizeX * nb_block_y * BlockSizeY * divide * sizeof(flottant);

  if(size > globalLeft) {
    fprintf(stderr, "# Exceeding GPU Memory Capacity \n");
    nb_block_y = nb_block_y / (size / globalLeft + 1) + 1;
    fprintf(stderr, "# Number of GPU iterations needed =  %lu\n", size / globalLeft + 1);
  }

  int XVectPerCall = nb_block_x * BlockSizeX;
  int YVectPerCall = nb_block_y * BlockSizeY;

  dim3 dimBlock(BlockSizeX, BlockSizeY, BlockSizeP);
  dim3 dimGrid(nb_block_x, nb_block_y, nb_block_p / nb_permut_per_th);

  SubTotalMem = XVectPerCall * YVectPerCall * divide * nb_permut_per_th * sizeof(flottant);
  fprintf(stderr, "# d_out memory allocated = %lu\n", SubTotalMem);

  size_t max_size = XVectPerCall * YVectPerCall * divide * nb_permut_per_th / 4;
  fprintf(stderr,"# Maximum number of values allowed per iteration = %G\n", (double)max_size);
  size_t sharedMem = ((VectorSize / 32) * 32) * sizeof(flottant);
  fprintf(stderr,"# SharedMem = %G\n", (double)sharedMem);		

  ///////////////// Allocation and Copy of input data ////////////////

  // allocate memory for arrays in the device
  hipMalloc( &dx_in, XSize);
  hipMalloc( &dy_in, YSize);
  hipMalloc( &dalpha, ASize);
  hipMalloc( &dp_in, PSize);
  hipMalloc( &cmpt, sizeof(int));
  hipMalloc( &flag, sizeof(int));
	
  //copy data from host to device
  hipMemcpyAsync(dx_in, x_in, XSize, hipMemcpyHostToDevice);
  hipMemcpyAsync(dy_in, y_in, YSize, hipMemcpyHostToDevice);
  hipMemcpyAsync(dalpha, alpha, ASize, hipMemcpyHostToDevice);
  hipMemcpyAsync(dp_in, p_python, PSize, hipMemcpyHostToDevice);
  hipMalloc(&d_out, SubTotalMem);
	
  ///////////////// kernel launches ///////////////// 

  int dof = nSubj - 1 - num_vector_Z;
  int pt_vector_X = 0, pt_vector_Y = 0;
  int loop_it = 1, nbXVect = 0, nbYVect = 0;

  while (pt_vector_X < num_vector_X) {
    nbXVect = XVectPerCall;
    if ( pt_vector_X + XVectPerCall > num_vector_X)
      nbXVect = num_vector_X - pt_vector_X;
    pt_vector_Y=0;
		
    while(pt_vector_Y < num_vector_Y) {
      nbYVect = YVectPerCall;
      if (pt_vector_Y + YVectPerCall > num_vector_Y)
	nbYVect = num_vector_Y - pt_vector_Y;

      fprintf(stderr, "# Iteration No %d, blocks: %dx%dx%d, threads: %dx%dx%d \n",
	      loop_it, dimGrid.x, dimGrid.y, dimGrid.z, dimBlock.x, dimBlock.y, dimBlock.z);

      hipMemset(d_out, 0 , SubTotalMem);
      hipMemset( cmpt, 0 , sizeof(int));
      hipMemset( flag, 0 , sizeof(int));
    
      //  hipFuncSetCacheConfig(reinterpret_cast<const void*>(Kernel_DotProd),hipFuncCachePreferL1);
      Kernel_DotProd<<< dimGrid , dimBlock, sharedMem>>>(&dx_in[pt_vector_X*VectorSize], 
							 &dy_in[pt_vector_Y*VectorSize], 
							 dp_in, d_out, dalpha, 
							 VectorSize, num_vector_Y,num_permut,
							 cmpt,dof,threshold, max_size, flag); 
      hipGetLastError();

      // retrieve counter and flag
      hipMemcpy(&hcmpt, cmpt, sizeof(int), hipMemcpyDeviceToHost);
      hipMemcpy(&hflag, flag, sizeof(int), hipMemcpyDeviceToHost);
      
      if(hflag!=0) {
	fprintf(stderr, "# WARNING: Beta size limit was reached. Some values were not calculated.\n");
	fprintf(stderr, "Try tuning the output size with the \"divide\" parameter.\n");
	hcmpt = max_size * 80 / 100;
      }
      pt_vector_Y += YVectPerCall;
      // retrieve scores for the iteration
      hipMemcpy(&beta[4 * total_cmpt], d_out, hcmpt * 4 * sizeof(flottant), hipMemcpyDeviceToHost);
      total_cmpt += hcmpt;
      loop_it++;
    }
    pt_vector_X += XVectPerCall;
  }

// cleanup
hipFree(dx_in);
hipFree(dalpha);
hipFree(dy_in);
hipFree(dp_in);
hipFree(d_out);

hipFree(cmpt);
hipFree(flag);

return total_cmpt;
}

/**
   Main for mulm regression
**/
int MULMRegression(flottant* X, int size_X,
		   flottant* Y, int size_Y,
		   flottant* alpha, int size_alpha,
		   int* P, int size_permut,
		   flottant* beta, int size_B,
		   int size_Z,
		   int VectorSize, int nSubj,
		   flottant divide,
		   double threshold,
		   int dev
		   )
{
  int num_vector_X = size_X / VectorSize;
  int num_vector_Y = size_Y / VectorSize;
  int num_vector_Z = size_Z / nSubj;
  int num_permut = size_permut / VectorSize;
  unsigned int values = 0;

  values = dotProdDevice(X,Y,beta,alpha,P,VectorSize, nSubj,
			 num_vector_X, num_vector_Y,num_vector_Z,num_permut,
			 divide,threshold,dev);

  return values;
}


